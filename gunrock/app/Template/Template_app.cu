// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file Template_app.cu
 *
 * @brief single-source shortest path (SSSP) application
 */

#include <gunrock/gunrock.h>
#include <gunrock/util/test_utils.cuh>
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

#include <gunrock/app/Template/Template_enactor.cuh>
#include <gunrock/app/Template/Template_test.cuh>

namespace gunrock {
namespace app {
namespace Template {

hipError_t UseParameters(util::Parameters &parameters)
{
    hipError_t retval = hipSuccess;
    GUARD_CU(UseParameters_app    (parameters));
    GUARD_CU(UseParameters_problem(parameters));
    GUARD_CU(UseParameters_enactor(parameters));

    // <todo>
    // GUARD_CU(parameters.Use<std::string>(
    //    "src",
    //    util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
    //    "0",
    //    "<Vertex-ID|random|largestdegree> The source vertices\n"
    //    "\tIf random, randomly select non-zero degree vertices;\n"
    //    "\tIf largestdegree, select vertices with largest degrees",
    //    __FILE__, __LINE__));
    // </todo>

    return retval;
}

struct RunParameters {

    // <todo> declare parameters, eg:
    // VertexT* srcs;
    // </todo>

    RunParameters() {}

    hipError_t Init(util::Parameters &parameters) {
        hipError_t retval = hipSuccess;
        
        // <todo> set parameters
        // </todo>
        
        return retval;
    }
    
    hipError_t SetRun(int run_num) {
        // This could do something like increment the srcs
    }
    
    hipError_t Release() {
        // Release parameters
    }
};

// ========================================= No todos below here =============================================

template <
    typename GraphT, 
    typename ValueT = typename GraphT::ValueT,
    typename VertexT = typename GraphT::VertexT,
    typename SizeT = typename GraphT::SizeT,
    typename ResultT
>
hipError_t Run(
    util::Parameters &parameters,
    GraphT           &graph,
    ResultT          &result,
    util::Location target = util::DEVICE)
{
    hipError_t retval = hipSuccess;
    
    typedef Problem<GraphT, RunParameters, ResultT> ProblemT;
    typedef Enactor<ProblemT> EnactorT;
    
    util::CpuTimer cpu_timer, total_timer;
    cpu_timer.Start(); total_timer.Start();
    
    // Init info
    util::Info info("Template", parameters, graph);

    // Problem specific parameters
    RunParameters run_parameters;
    run_parameters.Init(parameters);
    
    // Init problem
    ProblemT problem(parameters);
    GUARD_CU(problem.Init(graph, target));
    
    // Init enactor
    EnactorT enactor;
    GUARD_CU(enactor.Init(problem, target));
    
    cpu_timer.Stop();
    parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());
    
    int num_runs = parameters.Get<int >("num-runs");
    for (int run_num = 0; run_num < num_runs; ++run_num) {
        run_parameters.SetRun(run_num);
        
        GUARD_CU(problem.Reset(run_parameters, target));
        GUARD_CU(enactor.Reset(/*run_parameters,*/ target));
        
        cpu_timer.Start();
        GUARD_CU(enactor.Enact(/*run_parameters*/));
        cpu_timer.Stop();
        info.CollectSingleRun(cpu_timer.ElapsedMillis());
    }

    cpu_timer.Start();
    GUARD_CU(problem.Extract(result));

    // <todo> compute running statistics
    // !! This might be broken now
    // TODO: change NULL to problem specific per-vertex visited marker, e.g. h_distances
    // info.ComputeTraversalStats(enactor, (VertexT*)NULL);
    //Display_Memory_Usage(problem);
    // #ifdef ENABLE_PERFORMANCE_PROFILING
        //Display_Performance_Profiling(enactor);
    // #endif

    // Clean up
    run_parameters.Release();
    GUARD_CU(enactor.Release(target));
    GUARD_CU(problem.Release(target));
    cpu_timer.Stop(); total_timer.Stop();

    info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
    return retval;
}

} // namespace Template
} // namespace app
} // namespace gunrock
