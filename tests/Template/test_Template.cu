// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

#include <gunrock/app/Template/Template_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

// <todo>
namespace APP_NAMESPACE = app::Template;

const auto GRAPH_TYPES = graph::HAS_EDGE_VALUES | graph::HAS_CSR;

const auto SWITCH_TYPES = app::VERTEXT_U32B | app::VERTEXT_U64B |
        app::SIZET_U32B | app::SIZET_U64B |
        app::VALUET_U32B | app::DIRECTED | app::UNDIRECTED;

const bool SET_SRCS = false;

std::vector<std::string> switches{"advance-mode"};
// </todo>

// =============== No todos below this line ================

struct main_struct {
    template <typename VertexT, typename SizeT, typename ValueT>
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
        bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");
        
        typedef typename app::TestGraph<VertexT, SizeT, ValueT, GRAPH_TYPES> GraphT;
        typedef typename APP_NAMESPACE::Result<GraphT> ResultT;
        
        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        // Load graph
        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());
        
        if(SET_SRCS) {
            GUARD_CU(app::Set_Srcs(parameters, graph));
        }
        
        ResultT reference_results;
        if (!quick) {
            reference_results.Init(graph, parameters);
            reference_results.Reset();
            APP_NAMESPACE::CPU_Reference(graph, reference_results, quiet);
        }
            
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [reference_results, quick](util::Parameters &parameters, GraphT &graph) {
                hipError_t retval = hipSuccess;
                
                ResultT gunrock_results;
                gunrock_results.Init(graph, parameters);
                gunrock_results.Reset();
                
                APP_NAMESPACE::Run(parameters, graph, gunrock_results);
                
                if(!quick) {
                    APP_NAMESPACE::Validate_Results(gunrock_results, reference_results);   
                }
                
                gunrock_results.Release();
                return retval;
            }));
        
        reference_results.Release();
        return retval;
    }
};

int main(int argc, char** argv) {
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Template");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(APP_NAMESPACE::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help")) {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());
    
    return app::Switch_Types<SWITCH_TYPES>(parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
