// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/Template/Template_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters,
        VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
            graph::HAS_EDGE_VALUES | graph::HAS_CSR>
            GraphT;

        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph; // graph we process on

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        // force edge values to be 1, don't enable this unless you really want to
        //for (SizeT e=0; e < graph.edges; e++)
        //    graph.CsrT::edge_values[e] = 1;
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        // TODO: get srcs if needed, e.g.:
        GUARD_CU(app::Set_Srcs    (parameters, graph));
        int num_srcs = 0;

        // TODO: reference result on CPU, e.e.:
        // ValueT  **ref_distances = NULL;
        bool quick = parameters.Get<bool>("quick");
        // compute reference CPU SSSP solution for source-distance
        if (!quick)
        {
            bool quiet = parameters.Get<bool>("quiet");
            std::string validation = parameters.Get<std::string>("validation");
            util::PrintMsg("Computing reference value ...", !quiet);

            // TODO: get srcs if needed, e.g.:
            // std::vector<VertexT> srcs
            //    = parameters.Get<std::vector<VertexT> >("srcs");
            // num_srcs = srcs.size();

            // SizeT nodes = graph.nodes;
            // TODO: problem specific data, e.g.:
            // ref_distances = new ValueT*[num_srcs];
            for (int i = 0; i < num_srcs; i++)
            {
                // ref_distances[i] = new ValueT[nodes];
                // VertexT src = srcs[i];
                util::PrintMsg("__________________________", !quiet);
                float elapsed = app::Template::CPU_Reference(
                    graph.csr(),
                    // TODO: add problem specific data, e.g.:
                    // ref_distances[i], NULL, src,
                    quiet);
                util::PrintMsg("--------------------------\nRun "
                    + std::to_string(i) + " elapsed: "
                    + std::to_string(elapsed)
                    //+ " ms, src = " + std::to_string(src)
                    , !quiet);
            }
        }

        // TODO: add other switching parameters, if needed
        std::vector<std::string> switches{"advance-mode"};
        // TODO: add problem specific data
        GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
            [/*ref_distances*/](util::Parameters &parameters, GraphT &graph)
            {
                return app::Template::RunTests(parameters, graph/*, ref_distances*/);
            }));

        if (!quick)
        {
            // TODO: deallocate host references, e.g.:
            // for (int i = 0; i < num_srcs; i ++)
            // {
            //    delete[] ref_distances[i]; ref_distances[i] = NULL;
            // }
            // delete[] ref_distances; ref_distances = NULL;
        }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test Template");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::Template::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    // TODO: change available graph types, according to requirements
    return app::Switch_Types<
        app::VERTEXT_U32B | app::VERTEXT_U64B |
        app::SIZET_U32B | app::SIZET_U64B |
        app::VALUET_U32B | app::DIRECTED | app::UNDIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
