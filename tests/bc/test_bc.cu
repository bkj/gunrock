// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <iostream>
#include <gunrock/app/bc/bc_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_EDGE_VALUES | graph::HAS_CSR> GraphT; // PR different
        
        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        parameters.Set("undirected", true);

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        auto &graph_coo = graph.coo();
        for(VertexT i=0; i < graph_coo.edges; i++) {
            auto &edge_pair = graph_coo.edge_pairs[i];
            std::cout << edge_pair.x << "|" << edge_pair.y << std::endl;
        }
        
        // GUARD_CU(app::Set_Srcs    (parameters, graph));
        // int num_srcs = 0;

        // TODO: reference result on CPU, e.e.:
        // ValueT  **ref_distances = NULL;
        bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");
        
        if (!quick)
        {
            // std::string validation = parameters.Get<std::string>("validation");
            util::PrintMsg("Computing reference value ...", !quiet);

            // // TODO: get srcs if needed, e.g.:
            // std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT> >("srcs");
            // num_srcs = srcs.size();

            // SizeT nodes = graph.nodes;
            // // TODO: problem specific data, e.g.:
            // ref_distances = new ValueT*[num_srcs];
            // for (int i = 0; i < num_srcs; i++)
            // {
            //     // ref_distances[i] = new ValueT[nodes];
            //     // VertexT src = srcs[i];
            //     util::PrintMsg("__________________________", !quiet);
            //     float elapsed = app::Template::CPU_Reference(
            //         graph.csr(),
            //         // TODO: add problem specific data, e.g.:
            //         // ref_distances[i], NULL, src,
            //         quiet);
            //     util::PrintMsg("--------------------------\nRun "
            //         + std::to_string(i) + " elapsed: "
            //         + std::to_string(elapsed)
            //         //+ " ms, src = " + std::to_string(src)
            //         , !quiet);
            // }
        }

//         // TODO: add other switching parameters, if needed
//         std::vector<std::string> switches{"advance-mode"};
//         // TODO: add problem specific data
//         GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
//             [/*ref_distances*/](util::Parameters &parameters, GraphT &graph)
//             {
//                 return app::Template::RunTests(parameters, graph/*, ref_distances*/);
//             }));

//         if (!quick)
//         {
//             // TODO: deallocate host references, e.g.:
//             // for (int i = 0; i < num_srcs; i ++)
//             // {
//             //    delete[] ref_distances[i]; ref_distances[i] = NULL;
//             // }
//             // delete[] ref_distances; ref_distances = NULL;
//         }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test bc");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::bc::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    // TODO: change available graph types, according to requirements
    return app::Switch_Types<
        app::VERTEXT_U32B | // app::VERTEXT_U64B |
        app::SIZET_U32B | // app::SIZET_U64B |
        app::VALUET_F32B | 
        app::UNDIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
