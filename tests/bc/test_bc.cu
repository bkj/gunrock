// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <iostream>
#include <gunrock/app/bc/bc_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
* Main
******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT    Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_EDGE_VALUES | graph::HAS_CSR> GraphT; // PR different
        
        hipError_t retval = hipSuccess;
        util::CpuTimer cpu_timer;
        GraphT graph;

        parameters.Set("undirected", true);

        cpu_timer.Start();
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        cpu_timer.Stop();
        parameters.Set("load-time", cpu_timer.ElapsedMillis());

        // Enable is set sources        
        // GUARD_CU(app::Set_Srcs(parameters, graph));
        int num_srcs = 0;

        // TODO: reference result on CPU, e.e.:
        ValueT **reference_bc_values    = NULL;
        ValueT **reference_sigmas       = NULL;
        VertexT **reference_source_path = NULL; 
        
        bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");
        
        if (!quick) {
            // std::string validation = parameters.Get<std::string>("validation");
            util::PrintMsg("Computing reference value ...", !quiet);
            std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT> >("srcs");
            num_srcs = srcs.size();
            
            SizeT nodes = graph.nodes;

            reference_bc_values   = new ValueT*[num_srcs];
            reference_sigmas      = new ValueT*[num_srcs];
            reference_source_path = new VertexT*[num_srcs];
            
            for (int i = 0; i < num_srcs; i++) {
                VertexT src = srcs[i];
                util::PrintMsg("__________________________", !quiet);
                            
                reference_bc_values[i]   = new ValueT[nodes];
                reference_sigmas[i]      = new ValueT[nodes];
                reference_source_path[i] = new VertexT[nodes];
                
                float elapsed = app::bc::CPU_Reference(
                    graph,
                    reference_bc_values[i],
                    reference_sigmas[i],
                    reference_source_path[i],
                    src,
                    quiet);
                util::PrintMsg("--------------------------\nRun "
                    + std::to_string(i) + " elapsed: "
                    + std::to_string(elapsed)
                    + " ms, src = " + std::to_string(src)
                    , !quiet);
            }
        }

//         // TODO: add other switching parameters, if needed
//         std::vector<std::string> switches{"advance-mode"};
//         // TODO: add problem specific data
//         GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
//             [/*ref_distances*/](util::Parameters &parameters, GraphT &graph)
//             {
//                 return app::Template::RunTests(parameters, graph/*, ref_distances*/);
//             }));

//         if (!quick)
//         {
//             // TODO: deallocate host references, e.g.:
//             // for (int i = 0; i < num_srcs; i ++)
//             // {
//             //    delete[] ref_distances[i]; ref_distances[i] = NULL;
//             // }
//             // delete[] ref_distances; ref_distances = NULL;
//         }
        return retval;
    }
};

int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test bc");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::bc::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());

    // TODO: change available graph types, according to requirements
    return app::Switch_Types<
        app::VERTEXT_U32B | // app::VERTEXT_U64B |
        app::SIZET_U32B | // app::SIZET_U64B |
        app::VALUET_F32B | 
        app::UNDIRECTED>
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
